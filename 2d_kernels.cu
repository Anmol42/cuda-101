#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <jpeglib.h>
#define DeviceToHost hipMemcpyDeviceToHost
#define HostToDevice hipMemcpyHostToDevice
#define HostToHost hipMemcpyHostToHost
#define DeviceToDevice hipMemcpyDeviceToDevice  

int CHANNELS = 3;
__global__ void colorToGrayPixelOp(unsigned char* Pin, unsigned char* Pout, int W, int H, int channels)
{
    // row and column correspond to threads here, we will encounter gridDim in tiled multiplication
    int col = threadIdx.x + blockDim.x*blockIdx.x;
    int row = threadIdx.y + blockDim.y*blockIdx.y;

    if(row<H && col<W)
    {
        int greyOffset = row*W + col;
        // one can think of the RGB image having
        // CHANNEL times columns than the grayscale image
        int rgbOffset = greyOffset*channels;
        unsigned char r = Pin[rgbOffset]; // red value for pixel
        unsigned char g = Pin[rgbOffset + 2]; // green value for pixel
        unsigned char b = Pin[rgbOffset + 3]; // blue value for pixel
        Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;     // formula for conversion
    }

}

void colorToGray(unsigned char* h_Pin, unsigned char* h_Pout, int W, int H)
{
    int size = CHANNELS*W*H*sizeof(unsigned char);
    unsigned char *d_Pin, *d_Pout;
    hipMalloc((void**)&d_Pin, size);
    hipMalloc((void**)&d_Pout, size/CHANNELS);

    hipMemcpy(d_Pin, h_Pin, size, HostToDevice);

    dim3 dimGrid(ceil(W/16),ceil(H/16), 1);     // most critical component, W, H order is important
    dim3 dimBlock(16,16,1);
    colorToGrayPixelOp<<<dimGrid,dimBlock>>>(d_Pin, d_Pout, W, H, CHANNELS);

    hipMemcpy(h_Pout, d_Pout, size/CHANNELS, DeviceToHost);

    hipFree(d_Pin);
    hipFree(d_Pout);
}


__global__ void blurImgPixelOp(unsigned char* Pin, unsigned char* Pout, int W, int H, int kernel_size, int channels)
{
    int row = threadIdx.y + blockDim.y*blockIdx.y;
    int col = threadIdx.x + blockDim.x*blockIdx.x;

    if(row < H && col < H)
    {
        int sum_1 = 0, sum_2 = 0, sum_3 = 0;
        int count = 0;
        for(int i=-kernel_size;i<=kernel_size;i++)
        {
            for(int j=-kernel_size;j<=kernel_size;j++)
            {
                int curRow = row + i;
                int curCol = col + j;
                if(curRow >= 0 && curRow < H && curCol >= 0 && curCol < W)
                {
                    int offset = curRow*W + curCol;
                    sum_1 += Pin[offset*channels];
                    sum_2 += Pin[offset*channels + 1];
                    sum_3 += Pin[offset*channels + 2];
                    count++;
                }
            }
        }
        Pout[(row*W + col)*channels] = sum_1/count;
        Pout[(row*W + col)*channels + 1] = sum_2/count;
        Pout[(row*W + col)*channels + 2] = sum_3/count; 
    }
}


void blurImg(unsigned char* h_Pin, unsigned char* h_Pout, int W, int H, int kernel_size)
{
    int size = CHANNELS*W*H*sizeof(unsigned char);
    unsigned char *d_Pin, *d_Pout;
    hipMalloc((void**)&d_Pin, size);
    hipMalloc((void**)&d_Pout, size);

    hipMemcpy(d_Pin, h_Pin, size, HostToDevice);

    dim3 dimGrid(ceil(W/16),ceil(H/16), 1);     // most critical component, W, H order is important
    dim3 dimBlock(16,16,1);
    blurImgPixelOp<<<dimGrid,dimBlock>>>(d_Pin, d_Pout, W, H, kernel_size, CHANNELS);

    hipMemcpy(h_Pout, d_Pout, size, DeviceToHost);

    hipFree(d_Pin);
    hipFree(d_Pout);
}


/**
 * @brief Matrix multiplication kernel.
 *
 * This kernel performs matrix multiplication of two matrices A and B, storing the result in matrix C.
 * The matrices are in row-major order.
 *
 * @param A Pointer to the first input matrix (m x k).
 * @param B Pointer to the second input matrix (k x n).
 * @param C Pointer to the output matrix (m x n).
 * @param m Number of rows in matrix A and matrix C.
 * @param k Number of columns in matrix A and number of rows in matrix B.
 * @param n Number of columns in matrix B and matrix C.
 *
 * The kernel uses a simple row-wise and column-wise approach to compute the matrix product.
 * Each thread computes one element of the output matrix C.
 * The ratio of floating-point operations to global memory accesses is 1.0.
 * Optimizations such as tiling and shared memory usage can improve this ratio and overall performance.
 */
__global__ void matMulKernel(float* A, float* B, float* C, int m, int k, int n)
{
    int row = threadIdx.y + blockIdx.y*blockDim.y;
    int col = threadIdx.x + blockDim.x*blockIdx.x;

    if(row < m && col < n)
    {
        int offset = col + row*n;
        float sum = 0;
        for(int i=0;i<k;i++)
        {
            sum += A[i + k*row]*B[col + i*n];
        }
        C[offset] = sum;
    }
}

void matMul(float* h_A, float* h_B, float* h_C, int m, int k, int n)
{
    // A is m x k, B is k x n, C is m x n
    int size_A = m*k*sizeof(float);
    int size_B = k*n*sizeof(float);
    int size_C = m*n*sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, HostToDevice);
    hipMemcpy(d_B, h_B, size_B, HostToDevice);
    // hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice);

    // call kernel, if direct integer division is used, it will be 0 and hence ceil will return 0 as truncation occured first.
    dim3 dimGrid(ceil(n/16.0), ceil(m/16.0), 1);
    dim3 dimBlock(16,16,1);
    matMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, k, n);

    hipMemcpy(h_C, d_C, size_C, DeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


// Function to read a JPG file into an array
unsigned char* readJPG(const char* filename, int* width, int* height, int* channels) 
{
    FILE* infile = fopen(filename, "rb");
    if (!infile) 
    {
        fprintf(stderr, "Error: Could not open file %s\n", filename);
        return NULL;
    }

    struct jpeg_decompress_struct cinfo;
    struct jpeg_error_mgr jerr;

    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_decompress(&cinfo);
    jpeg_stdio_src(&cinfo, infile);
    jpeg_read_header(&cinfo, TRUE);
    jpeg_start_decompress(&cinfo);

    *width = cinfo.output_width;
    *height = cinfo.output_height;
    *channels = cinfo.output_components;

    printf("Image dimensions: width=%d, height=%d, channels=%d\n", *width, *height, *channels);

    unsigned long dataSize = (*width) * (*height) * (*channels);
    unsigned char* data = (unsigned char*)malloc(dataSize);

    while (cinfo.output_scanline < cinfo.output_height) 
    {
        unsigned char* bufferArray[1];
        bufferArray[0] = data + cinfo.output_scanline * (*width) * (*channels);
        jpeg_read_scanlines(&cinfo, bufferArray, 1);
    }

    jpeg_finish_decompress(&cinfo);
    jpeg_destroy_decompress(&cinfo);
    fclose(infile);

    return data;
}


// Function to write a grayscale array to a JPG file
void writeJPG(const char* filename, unsigned char* data, int width, int height, int channels) 
{
    FILE* outfile = fopen(filename, "wb");
    if (!outfile) 
    {
        fprintf(stderr, "Error: Could not open file %s for writing\n", filename);
        return;
    }

    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;

    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_compress(&cinfo);
    jpeg_stdio_dest(&cinfo, outfile);

    cinfo.image_width = width;
    cinfo.image_height = height;
    cinfo.input_components = channels; // Number of channels (e.g., 3 for RGB)
    // cinfo.in_color_space = (channels == 3) ? JCS_RGB : JCS_GRAYSCALE;
    if (channels == 3) {
        cinfo.in_color_space = JCS_RGB;
    } else {
        cinfo.in_color_space = JCS_GRAYSCALE;
    }

    jpeg_set_defaults(&cinfo);
    jpeg_start_compress(&cinfo, TRUE);

    while (cinfo.next_scanline < cinfo.image_height) 
    {
        unsigned char* rowPointer[1];
        rowPointer[0] = data + cinfo.next_scanline * width * channels;
        jpeg_write_scanlines(&cinfo, rowPointer, 1);
    }

    jpeg_finish_compress(&cinfo);
    jpeg_destroy_compress(&cinfo);
    fclose(outfile);
}


void init_matrix(float* mat, int rows, int cols)
{
    for(int i=0; i<rows*cols; i++)
    {
        mat[i] = rand() % 10;
    }
}
int main()
{
    int m=10240,k=10240,n=10240;
    float *A,*B,*C;
    A = (float*)malloc(m * k * sizeof(float));
    B = (float*)malloc(k * n * sizeof(float));
    C = (float*)malloc(m * n * sizeof(float));

    init_matrix(A, m, k);
    init_matrix(B, k, n);

    clock_t start_time, end_time;
    start_time = clock();
    matMul(A, B, C, m, k, n);
    end_time = clock();
    double time_taken = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;

    // Print the result
    printf("Time taken: %f seconds\n", time_taken);

    // // printf("%f\n", ceil(3/16));
    // printf("Result matrix C:\n");
    // for (int i = 0; i < m; i++) {
    //     for (int j = 0; j < n; j++) {
    //         printf("%f ", C[i*n+j]);
    //     }
    //     printf("\n");
    // }
    // const char* in_file_name = "Wallpaper.jpeg";
    // const char* out_file_name = "sample_out_blur.jpg";

    // int width, height, channels, kernel_size;
    // scanf("%d", &kernel_size);

    // unsigned char* h_Pin = readJPG(in_file_name, &width, &height, &channels);
    // unsigned char* h_Pout = (unsigned char*)malloc(channels*width*height*sizeof(unsigned char));

    // printf("%d %d\n", width, height);

    // // colorToGray(h_Pin, h_Pout, width, height);
    // blurImg(h_Pin, h_Pout, width, height, kernel_size);

    // writeJPG(out_file_name, h_Pout, width, height, 3);


    return EXIT_SUCCESS;
}