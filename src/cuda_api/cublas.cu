#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define D2D hipMemcpyDeviceToDevice
#define H2H hipMemcpyHostToHost

#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "cuBLAS error at %s:%d: %d\n", __FILE__, __LINE__, status); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


float* read_matrix_from_csv(const char* filename, int *rows, int *cols, char delimiter=',')
{
    FILE* file = fopen(filename, "r");
    if(file == NULL)
    {
        printf("Error: Unable to open file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    size_t capacity = 1024;  // Initial capacity for the buffer
    float* buffer = (float*)malloc(capacity * sizeof(float));
    if (!buffer) 
    {
        fprintf(stderr, "Error: Memory allocation failed\n");
        fclose(file);
        exit(EXIT_FAILURE);
    }

    size_t line_capacity = 1024; // Initial line buffer capacity
    char* line = (char*)malloc(line_capacity * sizeof(char));
    if (!line) 
    {
        fprintf(stderr, "Error: Memory allocation for line buffer failed\n");
        free(buffer);
        fclose(file);
        exit(EXIT_FAILURE);
    }

    *rows = 0;
    *cols = 0;
    int current_cols = 0;

    while (getline(&line, &line_capacity, file) != -1)  // Dynamically read each line
    {
        current_cols = 0;

        // Parse the line based on the delimiter
        char* token = strtok(line, &delimiter);
        while (token) 
        {
            if (*rows == 0) (*cols)++; // Count columns based on the first row

            if ((*rows) * (*cols) + current_cols >= capacity) 
            {
                capacity *= 2;
                buffer = (float*)realloc(buffer, capacity * sizeof(float)); // realloc copies the data for you
                if (!buffer) 
                {
                    fprintf(stderr, "Error: Memory reallocation failed\n");
                    free(line);
                    fclose(file);
                    exit(EXIT_FAILURE);
                }
            }

            buffer[(*rows) * (*cols) + current_cols] = strtof(token, NULL);
            current_cols++;
            token = strtok(NULL, &delimiter);
        }

        if (*rows > 0 && current_cols != *cols) 
        {
            fprintf(stderr, "Error: Inconsistent number of columns in row %d\n", *rows + 1);
            free(buffer);
            free(line);
            fclose(file);
            exit(EXIT_FAILURE);
        }

        (*rows)++;
    }

    fclose(file);
    free(line);

    // Resize buffer to the exact size
    buffer = (float*)realloc(buffer, (*rows) * (*cols) * sizeof(float));
    if (!buffer) 
    {
        fprintf(stderr, "Error: Memory reallocation failed\n");
        exit(EXIT_FAILURE);
    }

    return buffer;
}


int main()
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f, beta = 0.0f;
    return EXIT_SUCCESS;
}