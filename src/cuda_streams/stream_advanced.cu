#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define CHECK_CUDA_ERROR(call) \
    if ((call) != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(hipGetLastError())); \
        exit(EXIT_FAILURE); \
    }

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define D2D hipMemcpyDeviceToDevice
#define H2H hipMemcpyHostToHost


__global__ void kernel1(float *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] *= 2.0f;
    }
}

__global__ void kernel2(float *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] += 1.0f;
    }
}


void CUDART_CB myStreamCallback(hipStream_t stream, hipError_t status, void *userData) {
    printf("Stream callback: Operation completed\n");
}

//streams will be useful when processing inputs in batches for inferencing, for maximum utilization of GPU
int main()
{
    const int N = 1000000;
    size_t size = N * sizeof(float);
    float *h_data, *d_data;
    hipStream_t stream1, stream2;
    hipEvent_t event;
    // std::cout << event << std::endl;

    // Allocate host and device memory
    CHECK_CUDA_ERROR(hipHostMalloc(&h_data, size, hipHostMallocDefault));  // Pinned memory for faster transfers
    CHECK_CUDA_ERROR(hipMalloc(&d_data, size));

    // Initialize host data
    for (int i = 0; i < N; ++i) {
        h_data[i] = static_cast<float>(i);
    }

    // Create streams with different priorities
    int leastPriority, greatestPriority;
    CHECK_CUDA_ERROR(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));
    CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&stream1, hipStreamNonBlocking, leastPriority));
    CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&stream2, hipStreamNonBlocking, greatestPriority));

    // Create event
    CHECK_CUDA_ERROR(hipEventCreate(&event));

    // Copy data to device asynchronously
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_data, h_data, size, H2D, stream1));
    kernel1<<<(N + 255) / 256, 256, 0, stream1>>>(d_data, N);

    // Record event in stream1
    CHECK_CUDA_ERROR(hipEventRecord(event, stream1));

    // Make stream2 wait for event
    CHECK_CUDA_ERROR(hipStreamWaitEvent(stream2, event, 0));

    // Execute kernel in stream2
    kernel2<<<(N + 255) / 256, 256, 0, stream2>>>(d_data, N);

    // Add callback to stream2
    CHECK_CUDA_ERROR(hipStreamAddCallback(stream2, myStreamCallback, NULL, 0));

    // Asynchronous memory copy back to host
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_data, d_data, size, hipMemcpyDeviceToHost, stream2));

    // Synchronize streams
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream2));

    // Verify result
    for (int i = 0; i < N; ++i) {
        float expected = (static_cast<float>(i) * 2.0f) + 1.0f;
        if (fabs(h_data[i] - expected) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Clean up
    CHECK_CUDA_ERROR(hipHostFree(h_data));
    CHECK_CUDA_ERROR(hipFree(d_data));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream1));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream2));
    CHECK_CUDA_ERROR(hipEventDestroy(event));
    return EXIT_SUCCESS;
}