#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA_ERROR(call) \
    if ((call) != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(hipGetLastError())); \
        exit(EXIT_FAILURE); \
    }

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define D2D hipMemcpyDeviceToDevice
#define H2H hipMemcpyHostToHost

// Learning about streams through a basic Async copying of memory
__global__ void vectorAddKernel(float *A, float *B, float *C, int n)
{
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx < n)
    {
        C[idx] = A[idx] +  B[idx];
    }
}

void vectorAdd(float *A, float *B, float *C, int n)
{
    float *d_A, *d_B, *d_C;
    size_t size = n*sizeof(float);
    hipStream_t stream1, stream2;

    CHECK_CUDA_ERROR(hipMalloc((void**)&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_C, size));

    // Create streams
    CHECK_CUDA_ERROR(hipStreamCreate(&stream1));
    CHECK_CUDA_ERROR(hipStreamCreate(&stream2));

    // Copy inputs to device asynchronously
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_A, A, size, H2D, stream1));
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_B, B, size, H2D, stream2));

    int blockSize = 256;
    int gridSize = (n + blockSize - 1)/blockSize;   //ceil(n/blockSize)

    vectorAddKernel<<<gridSize, blockSize, 0, stream1>>>(d_A, d_B, d_C, n);

    // Copy result back to host asynchronously
    CHECK_CUDA_ERROR(hipMemcpyAsync(C, d_C, size, hipMemcpyDeviceToHost, stream2));

     // Synchronize streams
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream2));

    // Verify result
    for (int i = 0; i < n; ++i) {
        if (fabs(A[i] + B[i] - C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // Clean up
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream1));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream2));

}
int main() {
    // Your CUDA code here
    int n = 1<<20;
    float *A = (float*)malloc(n*sizeof(float));
    float *B = (float*)malloc(n*sizeof(float));
    float *C = (float*)malloc(n*sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < n; ++i) {
        A[i] = rand() / (float)RAND_MAX;
        B[i] = rand() / (float)RAND_MAX;
    }

    vectorAdd(A, B, C, n);
    return 0;
}