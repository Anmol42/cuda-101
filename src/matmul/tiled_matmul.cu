#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define DeviceToHost hipMemcpyDeviceToHost
#define HostToDevice hipMemcpyHostToDevice
#define HostToHost hipMemcpyHostToHost
#define DeviceToDevice hipMemcpyDeviceToDevice 



//Tiled matrix multiplication
__global__ void tiledMatMulKernel(float* A, float* B, float* result, int m, int k, int n)
{
    const int TILE_WIDTH = 64;
    // TILE_WIDTH must be known at compile time for this to work
    __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

    // int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    //global row and column is given here
    int row = threadIdx.y + blockDim.y*blockIdx.y;
    int col = threadIdx.x + blockDim.x*blockIdx.x;

    float value = 0;

    //load tile into shared memory
    for(int i=0;i<(k+TILE_WIDTH-1)/TILE_WIDTH;i++)
    {
        if(row < m && (TILE_WIDTH*i+tx) < k)
            Ads[ty][tx] = A[row*k+TILE_WIDTH*i+tx];
        else
            Ads[ty][tx] = 0.0f;
        if(col < n && (TILE_WIDTH*i+ty) < k)
            Bds[ty][tx] = B[col+n*(TILE_WIDTH*i+ty)];
        else
            Bds[ty][tx] = 0.0f;
        __syncthreads();

        for(int j=0;j<TILE_WIDTH;j++)
        {
            value += __fmaf_rn(Ads[ty][j],Bds[j][tx], 0.0f);
            __threadfence_block();
        }
        __syncthreads();
    }
    if(row<m && col<n)
        result[row*n+col] = value;

}


void matMul(float* h_A, float* h_B, float* h_C, int m, int k, int n)
{
    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    const int TILE_WIDTH = 64;
    // A is m x k, B is k x n, C is m x n
    int size_A = m*k*sizeof(float);
    int size_B = k*n*sizeof(float);
    int size_C = m*n*sizeof(float);

    float *d_A, *d_B, *d_C;
    hipEventRecord(start);
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken for memory allocation on GPU: %f ms\n", milliseconds);

    hipEventRecord(start);
    hipMemcpy(d_A, h_A, size_A, HostToDevice);
    hipMemcpy(d_B, h_B, size_B, HostToDevice);
    hipEventRecord(stop);

    // Memcpy is a sync function so calling a synchronise is not needed
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken for copying host data on GPU: %f ms\n", milliseconds);

    // call kernel, if direct integer division is used, it will be 0 and hence ceil will return 0 as truncation occured first.
    dim3 dimGrid(ceil(n/(float)(TILE_WIDTH)), ceil(m/(float)(TILE_WIDTH)), 1);
    dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);

    hipDeviceSynchronize();
    hipEventRecord(start);
   
    tiledMatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, k, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tiled MatMul kernel execution time: %f ms\n", milliseconds);
    double M = m, N = n, K = k;
    float total_operations = 2*M*N*K;
    float flops = total_operations / (milliseconds / 1000.0f);  // Total FLOPS
    float gflops = flops / 1e9;  // Convert to GFLOPS

    printf("Achieved GFLOPS: %f GFLOPS\n", gflops);

    double bytes_transferred = (2*M*N*K/(float)(TILE_WIDTH) + M*N)*sizeof(float);
    double achieved_bw = (bytes_transferred / (milliseconds / 1000.0f)) / 1e9;

    printf("Achieved Memory Bandwidth: %.2f GB/s, %f\n", achieved_bw, bytes_transferred);

    hipMemcpy(h_C, d_C, size_C, DeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


void init_matrix(float* mat, int rows, int cols)
{
    for(int i=0; i<rows*cols; i++)
    {
        mat[i] = -1000.0f + (static_cast<float>(rand()) / RAND_MAX) * 2000.0f;
    }
}


int main()
{
    srand(time(NULL));
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    
    int cores_per_sm = prop.multiProcessorCount * 128; //_ConvertSMVer2Cores(prop.major, prop.minor); we don't have the necessary helper header so hardcoding
    float clock_rate_ghz = prop.clockRate / 1e6f;  // Convert from kHz to GHz
    float theoretical_gflops = cores_per_sm * clock_rate_ghz * 2.0f;

    float mem_clock_ghz = prop.memoryClockRate / 1e6f;  // Convert kHz to GHz
    float mem_bus_width_bytes = prop.memoryBusWidth / 8.0f;  // Convert bits to bytes
    float theoretical_bw = mem_clock_ghz * mem_bus_width_bytes * 2;  // Factor 2 for DDR

    printf("Theoretical Peak GFLOPS: %.2f GFLOPS\n", theoretical_gflops);
    printf("Theoretical Memory Bandwidth: %.2f GB/s\n", theoretical_bw);

    int m=1024,k=1024,n=1024;
    float *A,*B,*C;
    A = (float*)malloc(m * k * sizeof(float));
    B = (float*)malloc(k * n * sizeof(float));
    C = (float*)malloc(m * n * sizeof(float));

    init_matrix(A, m, k);
    init_matrix(B, k, n);

    matMul(A, B, C, m, k, n);


    return  EXIT_SUCCESS;
}
