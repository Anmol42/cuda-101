#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define D2H hipMemcpyDeviceToHost
#define H2D hipMemcpyHostToDevice
#define H2H hipMemcpyHostToHost
#define D2D hipMemcpyDeviceToDevice 

#define TILE_WIDTH 32


/*
 *
 * This code implements a matrix multiplication using a coarser threading approach in CUDA.
 * By giving up a bit of parallelism from the tiled matrix multiplication, it aims to reduce 
 * redundant read accesses. This approach can help in optimizing memory access patterns and 
 * potentially improve performance by minimizing the number of redundant reads from global memory.
 *
 */


// Coarser matrix multiplication
__global__ void coarse_tiled_matmul_kernel(float* A, float* B, float* C, int m, int k, int n)
{
    int col1 = blockIdx.x*4*TILE_WIDTH + threadIdx.x*4;
    int col2 = blockIdx.x*4*TILE_WIDTH + threadIdx.x*4+1;
    int col3 = blockIdx.x*4*TILE_WIDTH + threadIdx.x*4+2;
    int col4 = blockIdx.x*4*TILE_WIDTH + threadIdx.x*4+3;
    int row = blockIdx.y*TILE_WIDTH + threadIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds1[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds2[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds3[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds4[TILE_WIDTH][TILE_WIDTH];
    double val1=0,val2=0,val3=0,val4=0;


    for(int i=0;i<(k+TILE_WIDTH-1)/TILE_WIDTH;i++)
    {
        if(row < m && i*TILE_WIDTH+tx < k)
            Ads[ty][tx] = A[row*k + i*TILE_WIDTH + tx];
        else Ads[ty][tx] = 0.0f;
        if(col1 < n && i*TILE_WIDTH + ty < k)
            Bds1[ty][tx] = B[col1 + n*(i*TILE_WIDTH + ty)];
        else Bds1[ty][tx] = 0.0f;
        if(col2 < n && i*TILE_WIDTH + ty < k)
            Bds2[ty][tx] = B[col2 + n*(i*TILE_WIDTH + ty)];
        else Bds2[ty][tx] = 0.0f;
        if(col3 < n && i*TILE_WIDTH + ty < k)
            Bds3[ty][tx] = B[col3 + n*(i*TILE_WIDTH + ty)];
        else Bds3[ty][tx] = 0.0f;
        if(col4 < n && i*TILE_WIDTH + ty < k)
            Bds4[ty][tx] = B[col4 + n*(i*TILE_WIDTH + ty)];
        else Bds4[ty][tx] = 0.0f;
        __syncthreads();
        
        
        for(int j=0; j<TILE_WIDTH;j++)
        {
            val1 += Ads[ty][j]*Bds1[j][tx];
            val2 += Ads[ty][j]*Bds2[j][tx];
            val3 += Ads[ty][j]*Bds3[j][tx];
            val4 += Ads[ty][j]*Bds4[j][tx];
        }
        __syncthreads();
    }
    if(row<m && col4<n)
    {
        C[row*n+col1] = val1;
        C[row*n+col2] = val2;
        C[row*n+col3] = val3;
        C[row*n+col4] = val4;
    }
    else if(row<m && col3<n)
    {
        C[row*n+col1] = val1;
        C[row*n+col2] = val2;
        C[row*n+col3] = val3;
    }
    else if(row<m && col2<n)
    {
        C[row*n+col1] = val1;
        C[row*n+col2] = val2;
    }
    else if(row<m && col1<n)
    {
        C[row*n+col1] = val1;
    }
}



float* read_matrix_from_csv(const char* filename, int *rows, int *cols, char delimiter=',')
{
    FILE* file = fopen(filename, "r");
    if(file == NULL)
    {
        printf("Error: Unable to open file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    size_t capacity = 1024;  // Initial capacity for the buffer
    float* buffer = (float*)malloc(capacity * sizeof(float));
    if (!buffer) 
    {
        fprintf(stderr, "Error: Memory allocation failed\n");
        fclose(file);
        exit(EXIT_FAILURE);
    }

    size_t line_capacity = 1024; // Initial line buffer capacity
    char* line = (char*)malloc(line_capacity * sizeof(char));
    if (!line) 
    {
        fprintf(stderr, "Error: Memory allocation for line buffer failed\n");
        free(buffer);
        fclose(file);
        exit(EXIT_FAILURE);
    }

    *rows = 0;
    *cols = 0;
    int current_cols = 0;

    while (getline(&line, &line_capacity, file) != -1)  // Dynamically read each line
    {
        current_cols = 0;

        // Parse the line based on the delimiter
        char* token = strtok(line, &delimiter);
        while (token) 
        {
            if (*rows == 0) (*cols)++; // Count columns based on the first row

            if ((*rows) * (*cols) + current_cols >= capacity) 
            {
                capacity *= 2;
                buffer = (float*)realloc(buffer, capacity * sizeof(float)); // realloc copies the data for you
                if (!buffer) 
                {
                    fprintf(stderr, "Error: Memory reallocation failed\n");
                    free(line);
                    fclose(file);
                    exit(EXIT_FAILURE);
                }
            }

            buffer[(*rows) * (*cols) + current_cols] = strtof(token, NULL);
            current_cols++;
            token = strtok(NULL, &delimiter);
        }

        if (*rows > 0 && current_cols != *cols) 
        {
            fprintf(stderr, "Error: Inconsistent number of columns in row %d\n", *rows + 1);
            free(buffer);
            free(line);
            fclose(file);
            exit(EXIT_FAILURE);
        }

        (*rows)++;
    }

    fclose(file);
    free(line);

    // Resize buffer to the exact size
    buffer = (float*)realloc(buffer, (*rows) * (*cols) * sizeof(float));
    if (!buffer) 
    {
        fprintf(stderr, "Error: Memory reallocation failed\n");
        exit(EXIT_FAILURE);
    }

    return buffer;
}


void matMulAccuracy(float *result, float* expected_result, int rows, int cols)
{
    int cnt = 0;
    float err = 1e-2;
    for(int i=0;i<rows;i++)
    {
        for(int j=0;j<cols;j++)
        {
            if(abs((result[i*cols+j] - expected_result[i*cols+j])/expected_result[i*cols+j]) > err)
            {
                cnt++;
                // printf("Error at index (%d, %d): %f != %f\n", i, j, result[i*cols+j], expected_result[i*cols+j]);
            }
            // else
            // {
            //     printf("Expected value matched at index (%d, %d): %f = %f\n", i, j, result[i*cols+j], expected_result[i*cols+j]);
            // }
        }
    }
    printf("Number of elements that differ by relative error more than %f: %d out of %ld\n", err, cnt, (long int)rows*cols);
}


void coarse_tiled_matmul(float* A, float* B, float* C, int m, int k, int n)
{
    hipEvent_t start, stop;
    float milliseconds;
    float *d_A, *d_B, *d_C;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t size_A = (long int)m*k*sizeof(float);
    size_t size_B = (long int)k*n*sizeof(float);
    size_t size_C = (long int)m*n*sizeof(float);

    hipEventRecord(start);
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken for memory allocation on GPU: %f ms\n", milliseconds);

    hipEventRecord(start);
    hipMemcpy(d_A, A, size_A, H2D);
    hipMemcpy(d_B, B, size_B, H2D);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken for copying host data on GPU: %f ms\n", milliseconds);

    dim3 dimGrid((n+4*TILE_WIDTH-1)/(4*TILE_WIDTH), (m+TILE_WIDTH-1)/TILE_WIDTH, 1);
    dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
    printf("Grid size: %dx%d\n", (n+4*TILE_WIDTH-1)/(4*TILE_WIDTH), (m+TILE_WIDTH-1)/TILE_WIDTH);


    hipDeviceSynchronize();
    hipEventRecord(start);
   
    coarse_tiled_matmul_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, k, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tiled MatMul kernel execution time: %f ms\n", milliseconds);
    double M = m, N = n, K = k;
    float total_operations = 2*M*N*K;
    float flops = total_operations / (milliseconds / 1000.0f);  // Total FLOPS
    float gflops = flops / 1e9;  // Convert to GFLOPS

    printf("Achieved GFLOPS: %f GFLOPS\n", gflops);

    // double bytes_transferred_real = (2*M*N*K/(float)(TILE_WIDTH) + M*N)*sizeof(float);
    // double achieved_bw = (bytes_transferred_real / (milliseconds / 1000.0f)) / 1e9;

    // printf("Achieved Memory Bandwidth: %.2f GB/s, %f\n", achieved_bw, bytes_transferred_real);

    hipMemcpy(C, d_C, size_C, D2H);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}
int main()
{
    srand(time(NULL));
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    
    int cores_per_sm = prop.multiProcessorCount * 128; //_ConvertSMVer2Cores(prop.major, prop.minor); we don't have the necessary helper header so hardcoding
    float clock_rate_ghz = prop.clockRate / 1e6f;  // Convert from kHz to GHz
    float theoretical_gflops = cores_per_sm * clock_rate_ghz * 2.0f;

    float mem_clock_ghz = prop.memoryClockRate / 1e6f;  // Convert kHz to GHz
    float mem_bus_width_bytes = prop.memoryBusWidth / 8.0f;  // Convert bits to bytes
    float theoretical_bw = mem_clock_ghz * mem_bus_width_bytes * 2;  // Factor 2 for DDR

    printf("Theoretical Peak GFLOPS: %.2f GFLOPS\n", theoretical_gflops);
    printf("Theoretical Memory Bandwidth: %.2f GB/s\n", theoretical_bw);

    int m,k,n;
    float *A,*B,*C,*actual_result;
    A = read_matrix_from_csv("./src/matmul/input_A_int.csv", &m, &k, ',');
    B = read_matrix_from_csv("./src/matmul/input_B_int.csv", &k, &n, ',');
    actual_result = read_matrix_from_csv("./src/matmul/output_int.csv", &m, &n);
    C = (float*)malloc(m * n * sizeof(float));

    printf("Matrix dimensions: A(%dx%d) B(%dx%d) C(%dx%d)\n",m,k,k,n,m,n);
    coarse_tiled_matmul(A,B,C,m,k,n);
    matMulAccuracy(C, actual_result, m, n);

    return EXIT_SUCCESS;
}